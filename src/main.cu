
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <cmath>
#include <ctime>
#include <iostream>
#include <random>
#include "hipblas.h"

#define A_MIN -5
#define A_MAX 5
#define PRINT false

__global__ void increseInV2byAbsV1Kernel(int n, float *V2, float *V1){
  // Linear index of the current thread
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  // Adding absolute values of all elements that lie on this thread
  while (idx < n) {
    V2[idx] += std::abs(V1[idx]);
    idx += blockDim.x * gridDim.x;
  }
}

float matrixLNormCUDA(const int m, const int n, float **a, const int blockSize){
  // Number of thread blocks in grid
  int gridSize = (int)ceil((float)n/blockSize);
  // Previous result ?
  float *hostV1, *hostV2;
  hostV1 = (float*)malloc(n*sizeof(float));
  hostV2 = (float*)malloc(n*sizeof(float));
  for(int j = 0; j < n; j++){
    hostV2[j] = 0.;
  }
  // Declaration of the device arrrays
  float *deviceV1, *deviceV2;
  hipMalloc(&deviceV1, n*sizeof(float));
  hipMalloc(&deviceV2, n*sizeof(float));
  // Copy host vectors to device
  hipMemcpy(deviceV2, hostV2, n*sizeof(float), hipMemcpyHostToDevice);
  // Loop over the all rows of the matrix A
  for(int i = 0; i < m; i++){
    // Filling hostV1 array with values of the current row of the matrix A
    for(int j = 0; j < n; j++){
      hostV1[j] = a[i][j];
    }
    // Copy hostV1 to deviceV1
    hipMemcpy(deviceV1, hostV1, n*sizeof(float), hipMemcpyHostToDevice);
    // Add absolute values of the current row to the deviceV2 array
    increseInV2byAbsV1Kernel<<<gridSize, blockSize>>>(n, deviceV2, deviceV1);
  }
  // Copy deviceV2 to host
  hipMemcpy(hostV2, deviceV2, n*sizeof(float), hipMemcpyDeviceToHost);
  // Finding maximum element of the deviceV2 array with the help of the CUBLAS
  hipblasHandle_t handle;
  hipblasStatus_t stat;
  hipblasCreate(&handle);
  int max_idx;
  stat = hipblasIsamax(handle, n, deviceV2, 1, &max_idx);
  if (stat != HIPBLAS_STATUS_SUCCESS)
    std::cout << "Max failed" << std::endl;
  hipblasDestroy(handle);
  // Result (CUBLAS indexates elements from 1)
  float result = hostV2[max_idx-1];
  // Release memory
  free(hostV2);
  hipFree(deviceV1);
  hipFree(deviceV2);
  // Return result
  return result;
}

int main(int argc, char **argv){
  // First point of the time measurement
  clock_t t = clock();
  // Shape of the matrix A (m x n)
  int m = std::atoi(argv[1]);
  int n = std::atoi(argv[2]);
  int blockSize = std::atoi(argv[3]);
  if (PRINT) std::cout << "m=" << m << ", n=" << n << ", blockSize=" << blockSize << std::endl;
  float **a; // Matrix A declaration
  a = new float *[m];
  for(int i = 0; i < m; i++)
    a[i] = new float[n];
  // Filling this matrix with random values  
  std::random_device rd;  // Will be used to obtain a seed for the random number engine
  std::mt19937 gen(rd()); // Standard mersenne_twister_engine seeded with rd()
  std::uniform_real_distribution<float> dis(A_MIN, A_MAX);
  if (PRINT) std::cout << "matrix A:" << std::endl;
  for(int i = 0; i < m; i++){
    for(int j = 0; j < n; j++){
      a[i][j] = int(dis(gen));
      if (PRINT) std::cout << a[i][j] << " ";
    }
    if (PRINT) std::cout << std::endl;
  }
  float temp_max = matrixLNormCUDA(m,n,a,blockSize);
  if (PRINT) std::cout << "result: " << temp_max << std::endl;
  // Last point of the time measurement
  if (PRINT) std::cout << "time [s]: ";
  std::cout << float(clock() - t) / float(CLOCKS_PER_SEC) << std::endl;
  return 0;
}
